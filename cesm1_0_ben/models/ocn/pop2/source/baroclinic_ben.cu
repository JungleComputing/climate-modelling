#include "hip/hip_runtime.h"
#include <stdio.h>

#include "timer.h"


createTimer(timer);

extern "C"
{
void print_ben(int n);
void ben_init(int id);
void ben_timer_start();
void ben_timer_end(char* name);
}

int rank;

__global__ void doNothing(float *nothing) {
	int tx = threadIdx.x;

	nothing += tx;
	
	*nothing = tx;
}


void print_ben(int n) {

	printf("PRINT_BEN IN C CALLED FROM FORTRAN! N=%d\n",n);
	
	float* d_nothing = 0;
	
	hipMalloc((void**) &d_nothing, 256*sizeof(float));
	
	dim3 threads(256, 1);
	dim3 grid(1,1);
	
	doNothing<<<grid, threads>>>(d_nothing);
	
	hipDeviceSynchronize();

	hipError_t err = hipGetLastError();
	printf("CUDA STATUS = %s\n", hipGetErrorString(err));
	
}


void ben_init(int id) {
	
	rank = id;

	
}


void ben_timer_start() {
	//if (rank == 0) {
		timer.reset();
		timer.start();
	//}
}



void ben_timer_end(char *name) {
	//if (rank == 0) {
		timer.stop();
		float ktime = timer.getTimeInMilliSeconds();
		
		printf("BEN Process %d: timer %s took %f ms\n",rank,name,ktime);
		timer.reset();
	//}
}
